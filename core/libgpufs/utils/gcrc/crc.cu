#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <time.h>
#include "crc.hpp"
#include "sse4crc.h"

#define DATA_SIZE		(1048576+0xf)
#define PAGE_SIZE		1024

double delta(timespec s, timespec e)
{
	double ds = s.tv_sec * 1e9 + s.tv_nsec;
	double de = e.tv_sec * 1e9 + e.tv_nsec;
	double d = de - ds;
	return d/1e6;
}


__global__ void pcrc32(const unsigned char* pbuf, size_t size, crc_t* ret)
{
	*ret = 0;
	__syncthreads();
	atomicXor(ret, dev_crc<UIntV4>(pbuf + PAGE_SIZE*blockIdx.x, PAGE_SIZE));
}

int main()
{
	char* ran = (char*)malloc(DATA_SIZE);
	for(int i = 0; i < DATA_SIZE; i++)
		ran[i] = random();
	unsigned char* gran;
	hipMalloc(&gran, DATA_SIZE);
	hipMemcpy(gran, ran, DATA_SIZE, hipMemcpyHostToDevice);
	crc_t* gret;
	hipMalloc(&gret, sizeof(crc_t));
	crc_t ret0, ret1, ret2, ret3;
	crc_t ret4;

	while (true) {
		timespec t0, t1, t2, t3, t4;
		timespec t5;
		timespec t6;
		timespec t7;

		clock_gettime(CLOCK_MONOTONIC, &t0);
		crc32b<<<1, 1>>>(gran, DATA_SIZE, gret);
		hipMemcpy(&ret0, gret, sizeof(crc_t), hipMemcpyDeviceToHost);

		clock_gettime(CLOCK_MONOTONIC, &t1);
		crc32t<UByte><<<1, 1>>>(gran, DATA_SIZE, gret);
		hipMemcpy(&ret1, gret, sizeof(crc_t), hipMemcpyDeviceToHost);

		clock_gettime(CLOCK_MONOTONIC, &t2);
		crc32t<UInt><<<1, 1>>>(gran, DATA_SIZE, gret);
		hipMemcpy(&ret2, gret, sizeof(crc_t), hipMemcpyDeviceToHost);

		clock_gettime(CLOCK_MONOTONIC, &t3);
		crc32t<UIntV4><<<1, 1>>>(gran, DATA_SIZE, gret);
		hipMemcpy(&ret3, gret, sizeof(crc_t), hipMemcpyDeviceToHost);

		clock_gettime(CLOCK_MONOTONIC, &t4);
		pcrc32<<<1, DATA_SIZE/PAGE_SIZE>>>(gran, DATA_SIZE, gret);
		hipMemcpy(&ret4, gret, sizeof(crc_t), hipMemcpyDeviceToHost);

		clock_gettime(CLOCK_MONOTONIC, &t5);
		crc_t cr = crc32(ran, DATA_SIZE, 0);
		clock_gettime(CLOCK_MONOTONIC, &t6);
		crc_t sse42cr = sse42_crc32(ran, DATA_SIZE, 0);
		clock_gettime(CLOCK_MONOTONIC, &t7);

		double d0 = delta(t0, t1);
		double d1 = delta(t1, t2);
		double d2 = delta(t2, t3);
		double d3 = delta(t3, t4);
		double d4 = delta(t4, t5);
		double d5 = delta(t5, t6);
		double d6 = delta(t6, t7);
		printf("crc32b         0x%X, time %f (ms), thpt %f MB/sec\n", ret0, d1,
				DATA_SIZE/1024.0/1024.0/(d0/1e3)
		);
		printf("crc32t<UByte>  0x%X, time %f (ms), thpt %f MB/sec\n", ret1, d1,
				DATA_SIZE/1024.0/1024.0/(d1/1e3)
		);
		printf("crc32t<UInt>   0x%X, time %f (ms), thpt %f MB/sec\n", ret2, d2,
				DATA_SIZE/1024.0/1024.0/(d2/1e3)
		);
		printf("crc32t<UIntV4> 0x%X, time %f (ms), thpt %f MB/sec\n", ret3, d3,
				DATA_SIZE/1024.0/1024.0/(d3/1e3)
		);
		printf("pcrc32t        0x%X, time %f (ms), thpt %f MB/sec\n", ret4, d4,
				DATA_SIZE/1024.0/1024.0/(d4/1e3)
		);
		printf("crc32          0x%X, time %f (ms), thpt %f MB/sec\n", cr, d5,
				DATA_SIZE/1024.0/1024.0/(d5/1e3)
		);
		printf("sse42crc32     0x%X, time %f (ms), thpt %f MB/sec\n", sse42cr, d6,
				DATA_SIZE/1024.0/1024.0/(d6/1e3)
		);
		printf("\n");
	}
}
